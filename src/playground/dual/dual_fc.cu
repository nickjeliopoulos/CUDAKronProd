#include "hip/hip_runtime.h"
#include "dual_fc.cuh"
#include "../cuda_helper.cuh"

namespace winter2024::dual{
	namespace {
		constexpr int32_t SM80_WARP_SIZE = 32;
		constexpr int32_t SM80_DUAL_PROBLEM_THREADS = 256;
		constexpr int32_t SM80_DUAL_PROBLEM_WARPS = SM80_DUAL_PROBLEM_THREADS / SM80_WARP_SIZE;
		constexpr int32_t SM80_DUAL_COMPUTE_B_CHUNKS_SIZE_ROWS = 1;
		constexpr int32_t SM80_DUAL_COMPUTE_B_CHUNKS_SIZE_COLS = SM80_DUAL_PROBLEM_THREADS;
	}


	__global__ void dual_conv1x1_fp32_cuda_kernel(
        const torch::PackedTensorAccessor64<float, 2, torch::RestrictPtrTraits> x, 
        const torch::PackedTensorAccessor64<float, 2, torch::RestrictPtrTraits> W, 
        const torch::PackedTensorAccessor64<float, 2, torch::RestrictPtrTraits> V, 
        torch::PackedTensorAccessor64<float, 2, torch::RestrictPtrTraits> C,
        float b,
        float c,
        int32_t B, 
        int32_t D_in, 
        int32_t D_out
    ) {
        // Each block handles one batch
        int32_t batch = blockIdx.x;
        // Each thread computes one output dimension.
        int32_t d_out = blockIdx.y * blockDim.x + threadIdx.x;

        // Allocate shared memory to store a tile of the x row.
        extern __shared__ float x_tile[];

        if (batch < B && d_out < D_out) {
            float left = b;
            float right = c;
            
            // Use the block dimension as the tile size.
            int32_t tile_size = blockDim.x;

            // Process the input in tiles.
            for (int32_t tile_start = 0; tile_start < D_in; tile_start += tile_size) {
                // Each thread loads one element of the tile.
                int32_t idx = tile_start + threadIdx.x;
                if (idx < D_in) {
                    x_tile[threadIdx.x] = x[batch][idx];
                } else {
                    x_tile[threadIdx.x] = 0;
                }
                __syncthreads();
                
                // Calculate the number of valid elements in this tile.
                int32_t current_tile_size = (D_in - tile_start < tile_size) ? (D_in - tile_start) : tile_size;

                // Iterate over the tile elements and update partial sums.
                for (int32_t i = 0; i < current_tile_size; i++) {
                    float x_val = x_tile[i];
                    left += x_val * W[tile_start + i][d_out];
                    right += x_val * V[tile_start + i][d_out];
                }
                __syncthreads();

            }
			
            C[batch][d_out] = left;
			C[batch][D_out + d_out] = right;
        }
    }


    torch::Tensor dual_fc(const torch::Tensor& x, 
                          const torch::Tensor& W, 
                          const torch::Tensor& V, 
                          const torch::Tensor& b, 
                          const torch::Tensor& c
						) {
        torch::Tensor C = torch::empty({x.size(0), 2 * W.size(1)}, x.options());
        int32_t B = x.size(0);
        int32_t D_in = x.size(1);
        int32_t D_out = W.size(1);

        dim3 threads(SM80_DUAL_PROBLEM_THREADS);
        dim3 blocks(B, (D_out + threads.x - 1) / threads.x);
        // Allocate shared memory equal to one tile (the size of one row segment of x)
        size_t sharedMemSize = SM80_DUAL_PROBLEM_THREADS * sizeof(float);

        dual_conv1x1_fp32_cuda_kernel<<<blocks, threads, sharedMemSize>>>(
            x.packed_accessor64<float, 2, torch::RestrictPtrTraits>(),
            W.packed_accessor64<float, 2, torch::RestrictPtrTraits>(),
            V.packed_accessor64<float, 2, torch::RestrictPtrTraits>(),
            C.packed_accessor64<float, 2, torch::RestrictPtrTraits>(),
            b.item<float>(), 
            c.item<float>(),
            B, 
            D_in, 
            D_out
        );
        
        return C;
    }
}
