#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>


int main() {
    hipDevice_t cuDevice;
    hipCtx_t cuContext;
    hipModule_t cuModule;
    hipFunction_t cuFunction;

    // Initialize CUDA driver API
    hipInit(0);
    
    // Get handle for device 0
    hipDeviceGet(&cuDevice, 0);

    // Create context
    hipCtxCreate(&cuContext, 0, cuDevice);

    // Load the module from the cubin file
    if (hipModuleLoad(&cuModule, "ptx_inspect_source.cubin") != hipSuccess) {
        std::cerr << "Failed to load module\n";
        return 1;
    }

    // Get the function handle from the module
    if (hipModuleGetFunction(&cuFunction, cuModule, "_ptx_simple_fmaf") != hipSuccess) {
        std::cerr << "Failed to get function\n";
        return 1;
    }

    // Prepare data
    float *d_input, *d_output;
    constexpr int size = 1024; // Example size, adjust as needed
	float h_input[size] = {0};
	float h_output[size] = {0};


    // Initialize host input data
    for (int i = 0; i < size; ++i) h_input[i] = static_cast<float>(i);

    // Allocate memory on device
    hipMalloc(&d_input, size * sizeof(float));
    hipMalloc(&d_output, size * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, h_input, size * sizeof(float), hipMemcpyHostToDevice);

    // Setup the kernel parameters
    void *args[] = { &d_input, &d_output };
    int blockSize = 256; // Example block size, adjust as needed
    int gridSize = (size + blockSize - 1) / blockSize;

    // Launch the kernel
    if (hipModuleLaunchKernel(cuFunction, 
                       gridSize, 1, 1,  // grid dim
                       blockSize, 1, 1, // block dim
                       0, NULL, args, NULL) != hipSuccess) {
        std::cerr << "Failed to launch kernel\n";
        return 1;
    }

    // Copy output back to host
    hipMemcpy(h_output, d_output, size * sizeof(float), hipMemcpyDeviceToHost);

	// Print output
	for (int i = 0; i < size; ++i) {
		std::cout << h_output[i] << " ";
	}

    // Clean up
    hipFree(d_input);
    hipFree(d_output);
    hipModuleUnload(cuModule);
    hipCtxDestroy(cuContext);

    return 0;
}